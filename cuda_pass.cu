
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <crypt.h>


//cuda encrypter
__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

__device__ bool cudaCmp(char* s1,char* s2){
    for(int i=0;i<sizeof(s1);i++){
        if(s1[i]!=s2[i])
            return false;
    }
    return true;

}

  //password checker
__device__ int check_pass(char* enc){
    int len_pass=10;


    char *passwords[]={
        "bwzdwy7323",
        "bwzdwy9523",
        "bwzdwy1662",
        "sorwqs7345",
        "gcfdwy4045",
        "bwzdwy7334",
        "bwzdwy7345",
        "zvydwy9562",
        "tpsdwy8453",
        "zvydwy4045"
    };
    
    for(int i=0;i<len_pass;i++){
        if(cudaCmp(passwords[i],enc))
            return 1;
        else
            return 0;
        
    }
    return 0;
}

//crack kernel
__global__ void crack_kernel(char *alphabet, char *number){
    char genRawPass[4];

    genRawPass[0]=alphabet[blockIdx.x];
    genRawPass[1]=alphabet[blockIdx.y];

    genRawPass[2]=number[threadIdx.x];
    genRawPass[3]=number[threadIdx.y];

    char *enc=CudaCrypt(genRawPass);
    if(check_pass(enc))
        printf("%s\n",genRawPass);
}

//main function
int main(int argc,char ** argv){
    char cpuAlphabet[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
    char cpuNumbers[26] = {'0','1','2','3','4','5','6','7','8','9'};

    char * gpuAlphabet;
    hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
    hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

    char * gpuNumbers;
    hipMalloc( (void**) &gpuNumbers, sizeof(char) * 26); 
    hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 26, hipMemcpyHostToDevice);

    clock_t begin = clock();
    crack_kernel<<<dim3(26,26,1),dim3(10,10,1)>>>(gpuAlphabet,gpuNumbers);
    hipDeviceSynchronize();
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Time Spend:%lf\n",time_spent);
    return 0;
}