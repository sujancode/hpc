#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"



__global__ void blur_image(unsigned char * gpu_imageOuput, unsigned char * gpu_imageInput,int width,int height){

    int counter=1;

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    
    int i=blockIdx.x;
    int j=threadIdx.x;


    int t_r=0;
	int t_g=0;
	int t_b=0;
    int t_a=0;


    if(i+1 && j-1){

        // int pos= idx/2-2;

        int pos=blockDim.x * (blockIdx.x+1) + threadIdx.x-1;
        int pixel = pos*4;
        
        t_r=gpu_imageInput[idx*4];
        t_g=gpu_imageInput[idx*4+1];
        t_b=gpu_imageInput[idx*4+2];
        t_a=gpu_imageInput[idx*4+3];

        t_r += gpu_imageInput[pixel];
        t_g += gpu_imageInput[1+pixel];
        t_b += gpu_imageInput[2+pixel];
        t_a += gpu_imageInput[3+pixel];
        
        counter++;



    }

    if(j+1){

        // int pos= idx/2-2;

        int pos=blockDim.x * (blockIdx.x) + threadIdx.x+1;

        int pixel = pos*4;

        t_r=gpu_imageInput[idx*4];
        t_g=gpu_imageInput[idx*4+1];
        t_b=gpu_imageInput[idx*4+2];
        t_a=gpu_imageInput[idx*4+3];

        t_r += gpu_imageInput[pixel];
        t_g += gpu_imageInput[1+pixel];
        t_b += gpu_imageInput[2+pixel];
        t_a += gpu_imageInput[3+pixel];

        counter++;
    }

    if(i+1 && j+1){

        // int pos= idx/2+1;

        int pos=blockDim.x * (blockIdx.x+1) + threadIdx.x+1;


        int pixel = pos*4;

        t_r=gpu_imageInput[idx*4];
        t_g=gpu_imageInput[idx*4+1];
        t_b=gpu_imageInput[idx*4+2];
        t_a=gpu_imageInput[idx*4+3];

        t_r += gpu_imageInput[pixel];
        t_g += gpu_imageInput[1+pixel];
        t_b += gpu_imageInput[2+pixel];
        t_a += gpu_imageInput[3+pixel];

        counter++;


    }

    if(i+1){
        // int pos= idx+1;

        int pos=blockDim.x * (blockIdx.x+1) + threadIdx.x;

        int pixel = pos*4;

        t_r=gpu_imageInput[idx*4];
        t_g=gpu_imageInput[idx*4+1];
        t_b=gpu_imageInput[idx*4+2];
        t_a=gpu_imageInput[idx*4+3];

        t_r += gpu_imageInput[pixel];
        t_g += gpu_imageInput[1+pixel];
        t_b += gpu_imageInput[2+pixel];
        t_a += gpu_imageInput[3+pixel];

        counter++;



    }

    if(j-1){

        // int pos= idx*2-2;
        int pos=blockDim.x * (blockIdx.x) + threadIdx.x-1;

        int pixel = pos*4;

        t_r=gpu_imageInput[idx*4];
        t_g=gpu_imageInput[idx*4+1];
        t_b=gpu_imageInput[idx*4+2];
        t_a=gpu_imageInput[idx*4+3];

        t_r += gpu_imageInput[pixel];
        t_g += gpu_imageInput[1+pixel];
        t_b += gpu_imageInput[2+pixel];
        t_a += gpu_imageInput[3+pixel];

        counter++;




    }

    if(i-1){

        // int pos= idx-1;
        int pos=blockDim.x * (blockIdx.x-1) + threadIdx.x;

        int pixel = pos*4;

        t_r=gpu_imageInput[idx*4];
        t_g=gpu_imageInput[idx*4+1];
        t_b=gpu_imageInput[idx*4+2];
        t_a=gpu_imageInput[idx*4+3];

        t_r += gpu_imageInput[pixel];
        t_g += gpu_imageInput[1+pixel];
        t_b += gpu_imageInput[2+pixel];
        t_a += gpu_imageInput[3+pixel];

        counter++;


    }
    
    int current_pixel=idx*4;

    gpu_imageOuput[current_pixel]=t_r/counter;
    gpu_imageOuput[1+current_pixel]=t_g/counter;
    gpu_imageOuput[2+current_pixel]=t_b/counter;
    gpu_imageOuput[3+current_pixel]=gpu_imageInput[3+current_pixel];


}



int main(int argc, char **argv){

	unsigned int error;
	unsigned int encError;
	unsigned char* image;
	unsigned int width;
	unsigned int height;
	const char* filename = "pic.png";
	const char* newFileName = "blur.png";

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	blur_image<<<height, width>>>(d_out, d_in,width,height);


	// copy back the result array to the CPU
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if(encError){
		printf("error %u: %s\n", error, lodepng_error_text(encError));
	}

	//free(image);
	//free(host_imageInput);
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
