/****************************************************************************
  Similar to factorise_3_0 but solves the problem with 4 threads using a 
  block method for search space partitioning. It is included here to 
  accompany a CUDA version of the program.

  Compile with:

   nvcc -o pswcuda pswcuda.cu
  
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__device__ int is_a_match(char *password){

	char pw1[]="BI1100";
	char pw2[]="KR2200";
	char pw3[]="RA4540";
	char pw4[]="PU6080";

	
	char *p1 = password;
	char *p2 = password;
	char *p3 = password;
	char *p4 = password;

	
	char *w1 = pw1;
	char *w2 = pw2;
	char *w3 = pw3;
	char *w4 = pw4;


	while(*p1 == *w1){


	if(*p1 == '\0'){

	printf("Password found : %s\n", pw1);
	return 1;

	}
	p1++;
	w1++;
}


while(*p2 == *w2){


	if(*p2 == '\0'){

	printf("Password found : %s\n", pw2);
	return 1;

	}
	p2++;
	w2++;
}


while(*p3 == *w3){


	if(*p3 == '\0'){

	printf("Password found : %s\n", pw3);
	return 1;

	}
	p3++;
	w3++;
}


while(*p4 == *w4){


	if(*p4 == '\0'){

	printf("Password found : %s\n", pw4);
	return 1;

	}
	p4++;
	w4++;
}
return 0;

}


__global__ void kernel(){

char alphabet[26] = {'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'S', 'T','U', 'V', 'W', 'X', 'Y', 'Z'};


char numbers[10] = {'0','1','2','3','4','5','6','7','8','9'};

char password[7];
password[6] = '\0';

int a, b, c, d;

for(a=0;a<10;a++){

	for(b=0;b<10;b++){

		for(c=0;c<10;c++){

			for(d=0;d<10;d++){

				password[0] = alphabet[blockIdx.x];
				password[1] = alphabet[threadIdx.x];
				password[2] = numbers[a];
				password[3] = numbers[b];
				password[4] = numbers[c];
				password[5] = numbers[d];

				if(is_a_match(password)){

					printf("Password found: %s\n", password);
			}
		}
			


	}



}

}




}



int time_difference(struct timespec *start, struct timespec *finish, 
                              long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main(int argc, char *argv[]) {
  struct timespec start, finish;   
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, &start);
 
	 kernel <<<26, 26>>>();
	hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", 
    time_elapsed, (time_elapsed/1.0e9)); 

  return 0;
}
